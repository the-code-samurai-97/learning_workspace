#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/device_reference.h>
#include <thrust/device_vector.h>
template <typename T>
void PrintThrustVector(const std::string& message,
                       thrust::device_vector<T>& list) {
  std::cout << message << '\n';
  int i{0};
  for (const auto& num : list) {
    std::cout << "list[" << i << "] = " << num << '\n';
    ++i;
  }
}
int main() {
  // From Raw Pointer to Device memory
  constexpr size_t N{10};
  int* raw_ptr;
  hipMalloc(&raw_ptr,
             N * sizeof(int));  // allocating in device in an array of 10
  thrust::device_ptr<int> device_ptr(
      raw_ptr);  // giving the starting address of raw pointer to device

  thrust::sequence(device_ptr, device_ptr + N, 4, 5);
  // Print the address and value of each array element on the device

  std::cout << " device_ptr\n";
  for (int i = 0; i < N; ++i) {
    std::cout << "device_ptr Address " << device_ptr + i << " device_ptr value "
              << device_ptr[i] << '\n';
  }
  std::cout << " device reference\n";

  for (int i{0}; i < N; ++i) {
    thrust::device_reference<int> device_reference = device_ptr[i];

    // Print the address and value of the device_reference
    std::cout << "device_reference Address " << &device_reference
              << " device_reference value " << device_reference << '\n';
  }

  // from device thrust to raw pointer
  thrust::device_ptr<int> dev_ptr =
      thrust::device_malloc<int>(N);  // cudamalloc + deviceptr
  thrust::sequence(dev_ptr, dev_ptr + N, 3, 6);

  int* raw_ptr_1 = new int[N];

  // Copy data from the device to the host
  thrust::copy(dev_ptr, dev_ptr + N, raw_ptr_1);

  // Print the address and value of each array element on the host
  for (int i = 0; i < N; ++i) {
    std::cout << "address " << &raw_ptr_1[i] << " value " << raw_ptr_1[i]
              << '\n';
  }

  delete[] raw_ptr_1;
  hipFree(raw_ptr);
  return 0;
}